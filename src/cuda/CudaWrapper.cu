#include "hip/hip_runtime.h"

# include "CudaWrapper.cuh"

namespace utils {

  CudaWrapper::CudaWrapper():
    CoreObject(std::string("wrapper")),

    m_propsLocker(),
    m_lastError()
  {
    setService("cuda");
  }

  bool
  CudaWrapper::launch(cuda::stream_t stream,
                      std::function<hipError_t(void)> func)
  {
    // Perform the call to the function.
    hipError_t err = func();

    if (isError(err)) {
      Guard guard(m_propsLocker);
      m_lastError = hipGetErrorString(err);

      return false;
    }

    // Wait for the stream to finish its task.
    return wait(stream);
  }

  cuda::stream_t
  CudaWrapper::create(bool* success) {
    // Attempt to create the stream.
    hipStream_t rawStream;

    hipError_t err = hipStreamCreateWithFlags(&rawStream, hipStreamNonBlocking);

    if (isError(err)) {
      Guard guard(m_propsLocker);
      m_lastError = hipGetErrorString(err);
    }

    // Fill the error status.
    if (success != nullptr) {
      *success = !isError(err);
    }

    return reinterpret_cast<cuda::stream_t>(rawStream);
  }

  bool
  CudaWrapper::wait(cuda::stream_t stream) {
    // Cast the input stream to usable data.
    hipStream_t rawStream = reinterpret_cast<hipStream_t>(stream);

    // Wait for the stream to terminate.
    hipError_t err = hipStreamSynchronize(rawStream);

    if (isError(err)) {
      Guard guard(m_propsLocker);
      m_lastError = hipGetErrorString(err);
    }

    return !isError(err);
  }

  bool
  CudaWrapper::destroy(cuda::stream_t stream) {
    // Attempt to destroy the stream.
    hipStream_t rawStream = reinterpret_cast<hipStream_t>(stream);

    hipError_t err = hipStreamDestroy(rawStream);

    if (isError(err)) {
      Guard guard(m_propsLocker);
      m_lastError = hipGetErrorString(err);
    }

    if (!isError(err)) {
      log("Destroyed cuda stream", utils::Level::Debug);
    }

    return !isError(err);
  }

}
