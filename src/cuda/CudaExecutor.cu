
# include "CudaExecutor.hh"
# include "mandelbulb_kernel.cuh"

namespace utils {

  CudaExecutor::CudaExecutor(unsigned size,
                             unsigned inElementSize,
                             const utils::Sizei& bufferSize,
                             unsigned outElementSize):
    CoreObject("executor"),

    m_poolLocker(),
    m_waiter(),
    m_poolRunning(false),
    m_jobsAvailable(false),

    m_threadsLocker(),
    m_threads(),
    m_cudaAPI(),
    m_schedulingData(),

    m_jobsLocker(),
    m_hPrioJobs(),
    m_nPrioJobs(),
    m_lPrioJobs(),
    m_batchIndex(0u),

    m_resultsLocker(),
    m_resultsHandling(false),
    m_results(),
    m_invalidateOld(true),
    m_resWaiter(),
    m_resultsThreadLocker(),
    m_resultsHandlingThread(),

    m_paramSize(),
    m_outElemSize(),

    onJobsCompleted()
  {
    setService("cuda");

    // Check consistency.
    if (size == 0u) {
      error(
        std::string("Could not create cuda executor service"),
        std::string("Invalid thread pool size of ") + std::to_string(size)
      );
    }

    // Create the scheduling data.
    createCudaSchedulingData(size, inElementSize, bufferSize, outElementSize);

    // Create the threads associated to this object.
    createThreadPool(size);
  }

  void
  CudaExecutor::notifyJobs() {
    // Protect from concurrent accesses.
    UniqueGuard guard(m_poolLocker);
    const std::lock_guard guard2(m_jobsLocker);

    // Determine whether some jobs have to be processed.
    if (!hasJobs()) {
      warn("Tried to start jobs processing but none are defined");
      return;
    }

    // Indicate that some jobs are available.
    m_jobsAvailable = true;

    // Notify working threads.
    m_waiter.notify_all();
  }

  void
  CudaExecutor::enqueueJobs(const std::vector<CudaJobShPtr>& jobs,
                            bool invalidate)
  {
    // Protect from concurrent accesses.
    const std::lock_guard guard(m_jobsLocker);

    // Invalidate jobs if needed: this include all the remaining jobs to process
    // but also notification about the ones currently being processed.
    if (invalidate) {
      m_hPrioJobs.clear();
      m_nPrioJobs.clear();
      m_lPrioJobs.clear();
    }

    {
      const std::lock_guard guard(m_resultsLocker);
      m_invalidateOld = invalidate;
    }

    // Build the job by providing the batch index for these jobs.
    for (unsigned id = 0u ; id < jobs.size() ; ++id) {
      // Consistency check.
      if (jobs[id] == nullptr) {
        warn("Discarding invalid null job " + std::to_string(id));
        continue;
      }

      std::vector<Job>* queue = nullptr;

      switch (jobs[id]->getPriority()) {
        case utils::Priority::High:
          queue = &m_hPrioJobs;
          break;
        case utils::Priority::Normal:
          queue = &m_nPrioJobs;
          break;
        case Priority::Low:
        default:
          // Assume low priority for unhandled priority.
          queue = &m_lPrioJobs;
          break;
      }

      if (queue == nullptr) {
        warn(
          "Could not find adequate queue for job \"" + jobs[id]->getName() + "\" with priority " +
          std::to_string(static_cast<int>(jobs[id]->getPriority()))
        );

        continue;
      }

      // Check whether the job matches the internal size both for
      // parameter and output result. If this is not the case we
      // won't be able to schedule it correctly.
      if (jobs[id]->getInputDataSize() != m_paramSize) {
        warn(
          std::string("Trying to submit job \"") + jobs[id]->getName() + "\" with a parameter size of " +
          std::to_string(jobs[id]->getInputDataSize()) + " while expected value is " + std::to_string(m_paramSize) +
          ", discarding it"
        );

        continue;
      }
      if (jobs[id]->getOutputDataSize() != m_outElemSize) {
        warn(
          std::string("Trying to submit job \"") + jobs[id]->getName() + "\" with a result size of " +
          std::to_string(jobs[id]->getOutputDataSize()) + " while expected value is " + std::to_string(m_outElemSize) +
          ", discarding it"
        );

        continue;
      }

      queue->push_back(
        Job{
          jobs[id],
          m_batchIndex
        }
      );
    }
  }

  void
  CudaExecutor::cancelJobs() {
    // Protect from concurrent accesses.
    UniqueGuard guard(m_poolLocker);
    const std::lock_guard guard2(m_jobsLocker);

    // Clear the internal queue so that no more jobs can be fetched.
    m_jobsAvailable = false;

    std::size_t count = m_hPrioJobs.size() + m_nPrioJobs.size() + m_lPrioJobs.size();
    verbose(
      "Clearing " + std::to_string(count) + " remaining job(s), next batch will be " +
      std::to_string(m_batchIndex)
    );

    m_hPrioJobs.clear();
    m_nPrioJobs.clear();
    m_lPrioJobs.clear();

    // Increment the batch index to mark any currently processing job
    // as invalid when it will complete.
    ++m_batchIndex;
  }

  void
  CudaExecutor::createThreadPool(unsigned size) {
    // Create the results handling thread.
    {
      const std::lock_guard guard(m_resultsLocker);
      m_resultsHandling = true;
    }
    {
      const std::lock_guard guard(m_resultsThreadLocker);
      m_resultsHandlingThread = std::thread(
        &CudaExecutor::resultsHandlingLoop,
        this
      );
    }

    // Start the pool.
    {
      UniqueGuard guard(m_poolLocker);
      m_poolRunning = true;
    }

    // Protect from concurrent creation of the pool.
    const std::lock_guard guard(m_threadsLocker);

    // Consistency check: verify that we can provide the
    // scheduling data to each thread.
    if (m_schedulingData.size() != size) {
      error(
        std::string("Could not create cuda executor service"),
        std::string("Should create ") + std::to_string(size) + " thread(s) but only " +
        std::to_string(m_schedulingData.size()) + " cuda stream(s) are available"
      );
    }

    m_threads.resize(size);
    for (unsigned id = 0u ; id < m_threads.size() ; ++id) {
      m_threads[id] = std::thread(
        &CudaExecutor::jobFetchingLoop,
        this,
        id,
        m_schedulingData[id]
      );
    }
  }

  void
  CudaExecutor::terminateThreads() {
    m_poolLocker.lock();

    // If no threads are created, nothing to do.
    if (!m_poolRunning) {
      m_poolLocker.unlock();
      return;
    }

    // Ask the threads to stop.
    m_poolRunning = false;
    m_poolLocker.unlock();
    m_waiter.notify_all();

    // Wait for all threads to finish.
    const std::lock_guard guard(m_threadsLocker);
    for (unsigned id = 0u ; id < m_threads.size() ; ++id) {
      m_threads[id].join();
    }

    m_threads.clear();

    // Now terminate the results handling thread.
    {
      m_resultsLocker.lock();

      // If the results thread is not started we don't have
      // to do anything.
      if (!m_resultsHandling) {
        m_resultsLocker.unlock();
        return;
      }

      // Stop the thread and wait for its termination.
      m_resultsHandling = false;
      m_resWaiter.notify_all();
      m_resultsLocker.unlock();

      const std::lock_guard guard3(m_resultsThreadLocker);
      m_resultsHandlingThread.join();
    }
  }

  void
  CudaExecutor::createCudaSchedulingData(unsigned count,
                                         unsigned paramSize,
                                         const utils::Sizei& bufferSize,
                                         unsigned elementSize)
  {
    // Protect from concurrent accesses to the threads' data.
    const std::lock_guard guard(m_threadsLocker);

    // Create resources for each needed thread.
    bool success = false;

    // Check consistency.
    if (paramSize == 0u) {
      error(
        std::string("Could not create cuda executor service"),
        std::string("Invalid parameter size of ") + std::to_string(paramSize)
      );
    }
    if (bufferSize.w() <= 0 || bufferSize.h() <= 0) {
      error(
        std::string("Could not create cuda executor service"),
        std::string("Invalid thread pool size of ") + bufferSize.toString()
      );
    }
    if (elementSize == 0u) {
      error(
        std::string("Could not create cuda executor service"),
        std::string("Invalid element size of ") + std::to_string(elementSize)
      );
    }

    // Register elements size.
    m_paramSize = paramSize;
    m_outElemSize = elementSize;

    unsigned id = 0u;
    while (id < count) {
      // Create the stream to use to schedule operations.
      cuda::stream_t stream = m_cudaAPI.create(&success);
      if (!success) {
        error(
          std::string("Could not create cuda executor service"),
          m_cudaAPI.getLastError()
        );
      }

      // Allocate the input parameters memory.
      void* paramMem = m_cudaAPI.allocate(m_paramSize, &success);
      if (!success) {
        error(
          std::string("Could not create cuda executor service"),
          m_cudaAPI.getLastError()
        );
      }

      // Allocate the output buffer.
      void* resBuffer = m_cudaAPI.allocate2D(bufferSize, elementSize, &success);
      if (!success || resBuffer == nullptr) {
        error(
          std::string("Could not create cuda executor service"),
          m_cudaAPI.getLastError()
        );
      }

      // Create the scheduling data and register it in the internal array.
      m_schedulingData.push_back(
        CudaSchedulingData{
          stream,
          paramMem,
          m_paramSize,
          resBuffer
        }
      );

      ++id;
    }
  }

  void
  CudaExecutor::destroyCudaSchedulingData() {
    // Protect from concurrent accesses to the threads' data.
    const std::lock_guard guard(m_threadsLocker);

    // Release memory for each created stream.
    for (unsigned id = 0u ; id < m_schedulingData.size() ; ++id) {
      CudaSchedulingData& d = m_schedulingData[id];

      // Destroy the stream.
      bool success = m_cudaAPI.destroy(d.stream);
      if (!success) {
        warn(
          std::string("Could not correctly destroy stream associated to thread ") + std::to_string(id) +
          " (error: \"" + m_cudaAPI.getLastError() + "\")"
        );
      }

      // Free the output buffer memory.
      success = m_cudaAPI.free(d.resBuffer);
      if (!success) {
        warn(
          std::string("Could not correctly destroy result buffer associated to thread ") + std::to_string(id) +
          " (error: \"" + m_cudaAPI.getLastError() + "\")"
        );
      }

      // Free the input parameters memory.
      success = m_cudaAPI.free(d.params);
      if (!success) {
        warn(
          std::string("Could not correctly destroy parameters buffer associated to thread ") + std::to_string(id) +
          " (error: \"" + m_cudaAPI.getLastError() + "\")"
        );
      }
    }
  }

  void
  CudaExecutor::jobFetchingLoop(unsigned threadID,
                                CudaSchedulingData gpuData)
  {
    verbose("Creating thread " + std::to_string(threadID) + " for thread pool");

    // Create the locker to use to wait for job to do.
    UniqueGuard tLock(m_poolLocker);

    while (m_poolRunning) {
      // Wait until either we are requested to stop or there are some
      // new jobs to process. Checking both conditions prevents us from
      // being falsely waked up (see spurious wakeups).
      m_waiter.wait(
        tLock,
        [&]() {
          return !m_poolRunning || m_jobsAvailable;
        }
      );

      // Check whether we need to process some jobs or exit the process.
      if (!m_poolRunning) {
        break;
      }

      // Attempt to retrieve a job to process.
      Job job = Job{nullptr, 0u};
      unsigned batch = 0u;
      std::size_t remaining = 0u;

      {
        const std::lock_guard guard(m_jobsLocker);

        // Fetch the highest priority job available.
        if (!m_hPrioJobs.empty()) {
          job = m_hPrioJobs.back();
          m_hPrioJobs.pop_back();
        }
        else if (!m_nPrioJobs.empty()) {
          job = m_nPrioJobs.back();
          m_nPrioJobs.pop_back();
        }
        else if (!m_lPrioJobs.empty()) {
          job = m_lPrioJobs.back();
          m_lPrioJobs.pop_back();
        }

        m_jobsAvailable = hasJobs();
        batch = m_batchIndex;

        remaining = m_hPrioJobs.size() + m_nPrioJobs.size() + m_lPrioJobs.size();
      }

      // Unlock the pool mutex so that we don't block other threads while
      // processing our chunk of job. This is what effectively allows for
      // concurrency.
      tLock.unlock();

      // If we could fetch something process it.
      if (job.task != nullptr) {
        verbose("Processing job for batch " + std::to_string(batch) + " in thread " + std::to_string(threadID) + " (remaining: " + std::to_string(remaining) + ")");

        // Execute the job and push it to the results array if it succeeded.
        if (scheduleAndExecute(*job.task, gpuData)) {
          // Notify the main thread about the result.
          UniqueGuard guard(m_resultsLocker);
          m_results.push_back(job);

          m_resWaiter.notify_one();
        }
      }

      // Once the job is done, reacquire the mutex in order to re-wait on
      // the condition variable.
      tLock.lock();
    }

    verbose("Terminating thread " + std::to_string(threadID) + " for scheduler pool");
  }

  void
  CudaExecutor::resultsHandlingLoop() {
    // Create the locker to use to wait for results to be processed.
    UniqueGuard rLock(m_resultsLocker);

    while (m_resultsHandling) {
      // Wait until either we are requested to stop or there are some
      // new results to analyze. Checking both conditions prevents us
      // from being falsely waked up (see spurious wakeups).
      m_resWaiter.wait(
        rLock,
        [&]() {
          return !m_resultsHandling || !m_results.empty();
        }
      );

      // Check whether we need to process some jobs or exit the process.
      if (!m_resultsHandling) {
        break;
      }

      // We want to notify listeners of the new results: to do that we
      // will copy the existing results to an internal handler, unlock
      // the mutex to allow for other results to be accumulated and
      // for longer interpretation processes to occur without ruining
      // the concurrency brought by the thread pool.
      std::vector<Job> local;
      local.swap(m_results);

      // Strip the batch index and keep only the jobs consistent with the
      // current one.
      std::vector<CudaJobShPtr> res;
      for (unsigned id = 0u ; id < local.size() ; ++id) {
        if (local[id].batch != m_batchIndex && m_invalidateOld) {
          verbose(
            std::string("Discarding job for old batch ") + std::to_string(local[id].batch) +
            " (current is " + std::to_string(m_batchIndex) + ")"
          );
          continue;
        }

        res.push_back(local[id].task);
      }

      // Notify listeners.
      rLock.unlock();
      onJobsCompleted.safeEmit(
        std::string("onJobsCompleted(") + std::to_string(res.size()) + ")",
        res
      );
      rLock.lock();
    }
  }

  bool
  CudaExecutor::scheduleAndExecute(CudaJob& job,
                                   CudaSchedulingData data)
  {
    // We need to first copy the input parameters of the job to device memory.
    bool success = m_cudaAPI.copyToDevice(data.stream, job.getInputData(), data.paramSize, data.params);
    if (!success) {
      warn(
        std::string("Could not copy parameter for job ") + job.getName() +
        "err: \"" + m_cudaAPI.getLastError() + "\")"
      );

      return false;
    }

    utils::Sizei size = job.getOutputSize();

    // Execute the job.
    success = m_cudaAPI.launch(
      data.stream,
      [&data, &size]() {
        hipStream_t cudaStream = reinterpret_cast<hipStream_t>(data.stream);

        // Use the mandelbulb wrapper to launch the kernel.
        return mandelbulb::mandelbulb_kernel_wrapper(
          cudaStream,
          data.params,
          data.resBuffer,
          size.w(),
          size.h()
        );
      }
    );
    if (!success) {
      warn(
        std::string("Could not launch job ") + job.getName() + ("err: \"") +
        m_cudaAPI.getLastError() + "\")"
      );

      return false;
    }

    // Wait for the job to complete.
    success = m_cudaAPI.wait(data.stream);
    if (!success) {
      warn(
        std::string("Job \"") + job.getName() + "\" failed (err: \"" +
        m_cudaAPI.getLastError() + "\")"
      );

      return false;
    }

    // Copy back the results to the job.
    success = m_cudaAPI.copyToHost2D(
      data.stream,
      job.getOutputSize(),
      job.getOutputDataSize(),
      data.resBuffer,
      job.getOutputData()
    );
    if (!success) {
      warn(
        std::string("Could not copy back result for job \"") + job.getName() +
        "\" (err: \"" + m_cudaAPI.getLastError() + "\")"
      );

      return false;
    }

    return true;
  }

}
